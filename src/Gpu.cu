#include "hip/hip_runtime.h"
#include "Gpu.h"

void GPU_test_current()
{
    int arraySize = 32;
    float data[arraySize] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};
    u_int8_t mask[(int)ceil(arraySize / 8.0f)] = { 0b10101010, 0b01010101, 0b10101010, 0b01010101};

    printf("Array to test: ");

	for (int i = 0 ; i < arraySize; i++){printf("%f ", data[i]);}

	printf("\n");

    float *d_data;
    float *d_data_box;
    char *d_mask;

    hipMalloc((void**)&d_data, arraySize * sizeof(float));
    hipMalloc((void**)&d_data_box, arraySize * sizeof(float));
    hipMalloc((void**)&d_mask, ceil(arraySize / 8.0f) * sizeof(char));

    hipMemcpy(d_data, data, arraySize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, ceil(arraySize / 8.0f) * sizeof(char), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    dim3 blockSize(2,1);
    dim3 gridSize(1,1);

    g_copyData_setMaskedScale1_removeBlanks<<<blockSize, gridSize>>>(d_data_box, d_data, d_mask, 16, 1, 2, 20);

    hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipMemcpy(data, d_data_box, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    printf("Array to test: ");

	for (int i = 0 ; i < arraySize; i++){printf("%f ", data[i]);}

	printf("\n");
}

void GPU_test_Gauss_X()
{
    float data[10] = {1,2,3,4,5,6,7,8,9,10};

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    float *d_data;
    float *d_data_box;

    hipMalloc((void**)&d_data, 10 * sizeof(float));
    hipMalloc((void**)&d_data_box, 10 * sizeof(float));

    hipMemset(d_data_box, 0, 10 * sizeof(float));

    hipMemcpy(d_data, data, 10 * sizeof(float), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipDeviceSynchronize();

    dim3 blockSize(8,1);
    dim3 gridSize(1,2);

    g_filter_gauss_X_flt<<<gridSize, blockSize, 29 * sizeof(float)>>>(d_data, 5, 1, 2, 1, 5);

    hipDeviceSynchronize();

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipMemcpy(data, d_data, 10 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    hipFree(d_data);
    hipFree(d_data_box);

}

void GPU_test_Gauss_Y()
{
    float data[10] = {1,2,3,4,5,6,7,8,9,10};

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    float *d_data;
    float *d_data_box;

    hipMalloc((void**)&d_data, 10 * sizeof(float));
    hipMalloc((void**)&d_data_box, 10 * sizeof(float));

    hipMemset(d_data_box, 0, 10 * sizeof(float));

    hipMemcpy(d_data, data, 10 * sizeof(float), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipDeviceSynchronize();

    dim3 blockSize(1,4);
    dim3 gridSize(5,1);

    g_filter_gauss_Y_flt<<<gridSize, blockSize, 29 * sizeof(float)>>>(d_data, 5, 2, 1, 1, 3);

    hipDeviceSynchronize();

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipMemcpy(data, d_data, 10 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    hipFree(d_data);
    hipFree(d_data_box);

}

void GPU_test_Boxcar_Z()
{
    float data[10] = {1,2,3,4,5,6,7,8,9,10};

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    float *d_data;
    float *d_data_box;

    hipMalloc((void**)&d_data, 10 * sizeof(float));
    hipMalloc((void**)&d_data_box, 10 * sizeof(float));

    hipMemset(d_data_box, 0, 10 * sizeof(float));

    hipMemcpy(d_data, data, 10 * sizeof(float), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipDeviceSynchronize();

    dim3 blockSize(1,1,2);
    dim3 gridSize(5,1);

    g_filter_boxcar_Z_flt<<<gridSize, blockSize, 7 * sizeof(float)>>>(d_data, 5, 1, 2, 1);

    hipDeviceSynchronize();

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipMemcpy(data, d_data, 10 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    hipFree(d_data);
    hipFree(d_data_box);

}

void GPU_test_sdt_dev(float *data, size_t size, size_t cadence, const int range)
{
    printf("Ref: %.3e\n", std_dev_val_flt(data, size, 0, cadence, range));

    float *d_data;
    float *d_data_box;

    hipMalloc((void**)&d_data, size * sizeof(float));
    hipMalloc((void**)&d_data_box, 2 * sizeof(float));

    hipMemset(d_data_box, 0, 2 * sizeof(float));

    hipMemcpy(d_data, data, size * sizeof(float), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipDeviceSynchronize();

    dim3 blockSize(1024);
    dim3 gridSize(1024);

    g_std_dev_val_flt<<<gridSize, blockSize, blockSize.x * 2 * sizeof(float)>>>(d_data, d_data_box, size, 0, cadence, range);

    hipDeviceSynchronize();

    g_std_dev_val_flt_final_step<<<1,1>>>(d_data_box);

    hipDeviceSynchronize();

    float noise[2] = {0,0};
    hipMemcpy(noise, d_data_box, 2 * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0 ; i < 2; i++){printf("noise: %.3e\n", noise[i]);;}

    hipFree(d_data);
    hipFree(d_data_box);
}

void GPU_test_median()
{
    float data[10] = {81,8,43,4,20,1,13,7,12,9};

    printf("Array to get median: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");

    float *d_data;
    float *d_data_box;

    hipMalloc((void**)&d_data, 10 * sizeof(float));
    hipMalloc((void**)&d_data_box, 10 * sizeof(float));

    hipMemset(d_data_box, 0, 10 * sizeof(float));

    hipMemcpy(d_data, data, 10 * sizeof(float), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));  
    }

    hipDeviceSynchronize();

    dim3 blockSize(2);
    dim3 gridSize(1);

    g_std_dev_val_flt<<<gridSize, blockSize, blockSize.x * 2 * sizeof(float)>>>(d_data, d_data_box, 10, 0, 4, 0);

    hipDeviceSynchronize();

    g_std_dev_val_flt_final_step<<<1,1>>>(d_data_box);

    hipDeviceSynchronize();

    g_DataCube_stat_mad_flt<<<gridSize, blockSize, blockSize.x * 14 * sizeof(float)>>>(d_data, d_data_box, 10, 1, 1, 0, 1, 0);

    hipDeviceSynchronize();

    hipMemcpy(data, d_data_box, 10 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("DataBox: ");

	for (int i = 0 ; i < 10; i++){printf("%f ", data[i]);}

	printf("\n");
}

void GPU_DataCube_filter_flt(char *data, char *maskdata, size_t data_size, const size_t *axis_size, const Array_dbl *kernels_spat, const Array_siz *kernels_spec, const double maskScaleXY, const double rms, const size_t cadence, const int range, const double threshold)
{
    printf("Starting GPU\n");

    // Error at start?
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));    
    }

    size_t n_iter;
    size_t radius_gauss;
    const double FWHM_CONST = 2.0 * sqrt(2.0 * log(2.0));

    size_t radius_boxcar;

    // check for CUDA capable device
    //hipFree(0);
    int deviceCount;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        exit(0);
    }

    size_t width = axis_size[0];
    size_t height = axis_size[1];
    size_t depth = axis_size[2];

    // Define memory on GPU
    float *d_data;
    float *d_data_box;
    float *d_data_duo;
    char *d_mask_data;
    char *d_original_mask;

    size_t free_bytes, total_bytes;
    hipError_t cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);

    if (cuda_status == hipSuccess) {
        printf("Total GPU Memory: %fMB\n", total_bytes / (1024.0f * 1024.0f));
        printf("Free GPU Memory: %fMB\n", free_bytes / (1024.0f * 1024.0f));
    } else {
        printf("hipMemGetInfo failed: %s\n", hipGetErrorString(cuda_status));
    }

    // Allocate and copy values from Host to Device
    printf("Allocating %fMB for the data\n", data_size * sizeof(float) / (1024.0f * 1024.0f));
    hipMalloc((void**)&d_data, data_size * sizeof(float));

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after malloc data: %s\n", hipGetErrorString(err));
        exit(1);
    }

    cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);

    if (cuda_status == hipSuccess) {
        printf("Total GPU Memory: %fMB\n", total_bytes / (1024.0f * 1024.0f));
        printf("Free GPU Memory: %fMB\n", free_bytes / (1024.0f * 1024.0f));
    } else {
        printf("hipMemGetInfo failed: %s\n", hipGetErrorString(cuda_status));
    }

    printf("Allocating %fMB for the box\n", data_size * sizeof(float) / (1024.0f * 1024.0f));
    hipMalloc((void**)&d_data_box, data_size * sizeof(float));

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after malloc box: %s\n", hipGetErrorString(err));
        exit(1);
    }

    cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);

    if (cuda_status == hipSuccess) {
        printf("Total GPU Memory: %fMB\n", total_bytes / (1024.0f * 1024.0f));
        printf("Free GPU Memory: %fMB\n", free_bytes / (1024.0f * 1024.0f));
    } else {
        printf("hipMemGetInfo failed: %s\n", hipGetErrorString(cuda_status));
    }

    printf("Allocating %fMB for the duo\n", 2 * sizeof(float) / (1024.0f * 1024.0f));
    hipMalloc((void**)&d_data_duo, 2 * sizeof(float));

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after malloc duo: %s\n", hipGetErrorString(err));
        exit(1);
    }

    cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);

    if (cuda_status == hipSuccess) {
        printf("Total GPU Memory: %fMB\n", total_bytes / (1024.0f * 1024.0f));
        printf("Free GPU Memory: %fMB\n", free_bytes / (1024.0f * 1024.0f));
    } else {
        printf("hipMemGetInfo failed: %s\n", hipGetErrorString(cuda_status));
    }

    printf("Allocating %fMB for the mask\n", data_size * sizeof(char) / (1024.0f * 1024.0f));
    hipMalloc((void**)&d_original_mask, data_size * sizeof(char));

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after malloc mask: %s\n", hipGetErrorString(err));
        exit(1);
    }

    cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);

    if (cuda_status == hipSuccess) {
        printf("Total GPU Memory: %fMB\n", total_bytes / (1024.0f * 1024.0f));
        printf("Free GPU Memory: %fMB\n", free_bytes / (1024.0f * 1024.0f));
    } else {
        printf("hipMemGetInfo failed: %s\n", hipGetErrorString(cuda_status));
    }

    hipMemcpy(d_data, data, data_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data_box, data, data_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_original_mask, maskdata, data_size * sizeof(char), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after copy: %s\n", hipGetErrorString(err));    
    }

    // Set up Bitmask space efficient on GPU one byte handles 8 entries in from the cube,
    // since here we only need to mask pixels, not differ between individual sources
    size_t d_mask_size = ceil(width / 8.0f) * height * depth * sizeof(char);
	hipMalloc((void**)&d_mask_data, d_mask_size);
    hipMemset(d_mask_data, 0, d_mask_size);

    //GPU_DataCube_copy_mask_8_to_1(d_mask_data, d_original_mask, axis_size);


    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after seting up mask data: %s\n", hipGetErrorString(err));    
    }

    dim3 blockSizeMS(32,32);
    dim3 gridSizeMS((width + blockSizeMS.x - 1) / blockSizeMS.x,
                    (height + blockSizeMS.y - 1) / blockSizeMS.y);

    dim3 blockSizeX(1024,1);
    dim3 gridSizeX(1,height);

    dim3 blockSizeY(1,1024);
    dim3 gridSizeY(width,1);

    dim3 blockSizeZ(1024,1);
    dim3 gridSizeZ(width,1);

    dim3 blockSizeNoise(1024);
    dim3 gridSizeNoise(1024);

    for(size_t i = 0; i < Array_dbl_get_size(kernels_spat); ++i)
	{
        for(size_t j = 0; j < Array_siz_get_size(kernels_spec); ++j)
		{
            hipMemset(d_data_duo, 0, 2 * sizeof(float));

            if (Array_dbl_get(kernels_spat, i) || Array_siz_get(kernels_spec, j))
            {
                optimal_filter_size_dbl(Array_dbl_get(kernels_spat, i) / FWHM_CONST, &radius_gauss, &n_iter);
                radius_boxcar = Array_siz_get(kernels_spec, j) / 2;

                printf("[%.1f] x [%lu]\n", Array_dbl_get(kernels_spat, i), Array_siz_get(kernels_spec, j));

                // Copy maskScaled data from d_data to d_data_box and replace blanks
                if (maskScaleXY >= 0.0)
                {
                    printf("Starting Kernels\n");

                    g_copyData_setMaskedScale8_removeBlanks<<<gridSizeMS, blockSizeMS>>>(d_data_box, d_data, d_original_mask, width, height, depth, maskScaleXY * rms);
                    hipDeviceSynchronize();

                    err = hipGetLastError();
                    if (err != hipSuccess)
                    {
                        printf("Cuda error after Mask Kernel: %s\n", hipGetErrorString(err));    
                    }
                }
                else
                {
                    g_copyData_removeBlanks<<<gridSizeMS, blockSizeMS>>>(d_data_box, d_data, width, height, depth);
                    hipDeviceSynchronize();
                }
                
                if(radius_gauss > 0)
                {
                    printf("Launching Gauss X…\n");

                    g_filter_gauss_X_flt<<<gridSizeX, blockSizeX, (radius_gauss * 3 + width * 2) * sizeof(float)>>>(d_data_box, width, height, depth, radius_gauss, n_iter);
                    hipDeviceSynchronize();

                    err = hipGetLastError();
                    if (err != hipSuccess)
                    {
                        printf("Cuda error after X Kernel: %s\n", hipGetErrorString(err));    
                    }
                }
                    
                if(radius_gauss > 0)
                {
                    printf("Launching Gauss Y…\n");

                    g_filter_gauss_Y_flt<<<gridSizeY, blockSizeY, (radius_gauss * 3 + height * 2) * sizeof(float)>>>(d_data_box, width, height, depth, radius_gauss, n_iter);
                    hipDeviceSynchronize();

                    err = hipGetLastError();
                    if (err != hipSuccess)
                    {
                        printf("Cuda error after Y Kernel: %s\n", hipGetErrorString(err));    
                    }
                }

                if(radius_boxcar > 0) 
                {
                    printf("Launching Boxcar Z…\n");

                    g_filter_boxcar_Z_flt<<<gridSizeZ, blockSizeZ, (radius_boxcar * 3 + depth * 2) * sizeof(float)>>>(d_data_box, width, height, depth, radius_boxcar);
                    hipDeviceSynchronize();

                    err = hipGetLastError();
                    if (err != hipSuccess)
                    {
                        printf("Cuda error after Z Kernel: %s\n", hipGetErrorString(err));    
                    }
                }

                printf("Starting Kernels for Mask\n");
                g_addBlanks<<<gridSizeMS, blockSizeMS>>>(d_data_box, d_data, width, height, depth);

                g_std_dev_val_flt<<<gridSizeNoise, blockSizeNoise, 1024 * 2 * sizeof(float)>>>(d_data_box, d_data_duo, data_size, 0.0f, cadence, range);
                hipDeviceSynchronize();

                float noise[2] = {0,0};
                hipMemcpy(noise, d_data_duo, 2 * sizeof(float), hipMemcpyDeviceToHost);

                printf("noise: %.3e\n", noise[0]);
                printf("Count: %.3e\n", noise[1]);

                g_std_dev_val_flt_final_step<<<1,1>>>(d_data_duo);
                hipDeviceSynchronize();

                hipMemcpy(noise, d_data_duo, 2 * sizeof(float), hipMemcpyDeviceToHost);

                printf("Final noise: %.3e\n\n", noise[0]);

                g_Mask8<<<gridSizeMS, blockSizeMS>>>(d_data_box, d_original_mask, width, height, depth, threshold, d_data_duo, 1);

                err = hipGetLastError();
                if (err != hipSuccess)
                {
                    printf("Cuda error after noise calc: %s\n", hipGetErrorString(err));
                }
            }
            else
            {
                g_std_dev_val_flt<<<gridSizeNoise, blockSizeNoise, 1024 * 2 * sizeof(float)>>>(d_data, d_data_duo, data_size, 0.0f, cadence, range);
                hipDeviceSynchronize();
                g_std_dev_val_flt_final_step<<<1,1>>>(d_data_duo);
                hipDeviceSynchronize();

                float noise[2] = {0,0};
                hipMemcpy(noise, d_data_duo, 2 * sizeof(float), hipMemcpyDeviceToHost);

                printf("noise: %.3e\n\n", noise[0]);

                g_Mask8<<<gridSizeMS, blockSizeMS>>>(d_data, d_original_mask, width, height, depth, threshold, d_data_duo, 1);

                err = hipGetLastError();
                if (err != hipSuccess)
                {
                    printf("Cuda error after noise calc: %s\n", hipGetErrorString(err));    
                }
            }
        }
    }

    hipMemcpy(maskdata, d_original_mask, data_size * sizeof(char), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_data);
    hipFree(d_data_box);
    hipFree(d_mask_data);
    hipFree(d_original_mask);
    hipFree(d_data_duo);
}

__global__ void g_copyData_removeBlanks(float *data_box, float *data, const size_t width, const size_t height, const size_t depth)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {return;}

    size_t index = x + y * width;

    while (index < width * height * depth)
    {
        data_box[index] = FILTER_NAN(data[index]);
        index += width * height;
    }
}

__global__ void g_addBlanks(float *data_box, float* data, const size_t width, const size_t height, const size_t depth)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {return;}

    size_t index = x + y * width;

    while (index < width * height * depth)
    {
        if(IS_NAN(data[index])) data_box[index] = NAN;
        index += width * height;
    }
}

__global__ void g_copyData_setMaskedScale1_removeBlanks(float *data_box, float *data, char *maskData1, const size_t width, const size_t height, const size_t depth, const float value)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t maskWidth = ceil(width / 8.0f);
    size_t index = y * maskWidth + x;
    size_t page_size = width * height;
    size_t mask_page_size = maskWidth * height;

    if (x >= maskWidth || y >= height) {return;}

    size_t pageNumber = 0;
    while (index < mask_page_size * depth)
    {
        if (x == maskWidth - 1)
        {
            for (int i = 0; x * 8 + i < width; ++i)
            {
                data_box[index * 8 + i] = (*(maskData1 + index) & (1 << (7 - i))) ? copysign(value, data[index * 8 + i]) : data[index * 8 + i];
            }
        }
        else
        {
            data_box[x * 8 + y * width + 0] = (*(maskData1 + index) & (1 << 7)) ? copysign(value, data[x * 8 + y * width + 0]) : FILTER_NAN(data[x * 8 + y * width + 0]);
            data_box[x * 8 + y * width + 1] = (*(maskData1 + index) & (1 << 6)) ? copysign(value, data[x * 8 + y * width + 1]) : FILTER_NAN(data[x * 8 + y * width + 1]);
            data_box[x * 8 + y * width + 2] = (*(maskData1 + index) & (1 << 5)) ? copysign(value, data[x * 8 + y * width + 2]) : FILTER_NAN(data[x * 8 + y * width + 2]);
            data_box[x * 8 + y * width + 3] = (*(maskData1 + index) & (1 << 4)) ? copysign(value, data[x * 8 + y * width + 3]) : FILTER_NAN(data[x * 8 + y * width + 3]);
            data_box[x * 8 + y * width + 4] = (*(maskData1 + index) & (1 << 3)) ? copysign(value, data[x * 8 + y * width + 4]) : FILTER_NAN(data[x * 8 + y * width + 4]);
            data_box[x * 8 + y * width + 5] = (*(maskData1 + index) & (1 << 2)) ? copysign(value, data[x * 8 + y * width + 5]) : FILTER_NAN(data[x * 8 + y * width + 5]);
            data_box[x * 8 + y * width + 6] = (*(maskData1 + index) & (1 << 1)) ? copysign(value, data[x * 8 + y * width + 6]) : FILTER_NAN(data[x * 8 + y * width + 6]);
            data_box[x * 8 + y * width + 7] = (*(maskData1 + index) & (1 << 0)) ? copysign(value, data[x * 8 + y * width + 7]) : FILTER_NAN(data[x * 8 + y * width + 7]);
        }
        
        index += mask_page_size;
        pageNumber++;
    }
}

__global__ void g_copyData_setMaskedScale8_removeBlanks(float *data_box, float *data, char *maskData8, const size_t width, const size_t height, const size_t depth, const float value)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {return;}

    size_t index = x + y * width;

    while (index < width * height * depth)
    {
        data_box[index] = ((int8_t)(maskData8[index])) ? copysign(value, data[index]) : FILTER_NAN(data[index]);
        index += width * height;
    }
}

__global__ void g_maskScale_remove_blanks_flt(float *data, char *mask, const size_t width, const size_t height, const size_t depth, const float value)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t mask_width = ceil(width / 8.0f);
    size_t mask_slice_size = mask_width * height;

    if (x < width && y < height)
    {
        for (int z = 0; z < depth; ++z)
        {
            size_t maskIndex = z * mask_slice_size + y * mask_width + x / 8;
            size_t index = z * width * height + y * width + x;
            data[index] = data[index] * (1 - (mask[maskIndex] >> (7 - (x%8))) & 1) + value * ((mask[maskIndex] >> (7 - (x%8))) & 1);
            
        }
    }
}

__global__ void g_filter_gauss_X_flt(float *data, const size_t width, const size_t height, const size_t depth, const size_t radius, const size_t n_iter)
{
    size_t x = threadIdx.x;
    size_t y = blockIdx.y;

    extern __shared__ float s_data_GX_flt[];
    float *s_data_src = s_data_GX_flt + radius;
    float *s_data_dst = s_data_GX_flt + 2 * radius + width;

    if (x == 0) 
    {
        for (int i = radius; i--;)
        {
            *(s_data_GX_flt + i) = *(s_data_GX_flt + radius + width + i) = *(s_data_GX_flt + 2 * radius + 2 * width + i) = 0.0f;
        }
    }

    for (int z = 0; z < depth; ++z)
    {
        //inline size_t index = x + y * width + z * width * height;
        while(x < width && y < height)
        {
            *(s_data_src + x) = data[x + y * width + z * width * height];
            x += blockDim.x;
        }

        x = threadIdx.x;
        __syncthreads();

        for (int n = n_iter; n--;)
        {
            while(x < width && y < height)
            {
                *(s_data_dst + x) = *(s_data_src + x);
                for (int i = radius; i--;)
                {
                    *(s_data_dst + x) += *(s_data_src + x + (i + 1)) + *(s_data_src + x - (i + 1));
                }
                *(s_data_dst + x) /= 2 * radius + 1;
                x += blockDim.x;
            }

            x = threadIdx.x;
            float *tmp = s_data_src;
            s_data_src = s_data_dst;
            s_data_dst = tmp;
            __syncthreads();
        }

        while(x < width && y < height)
        {
            data[x + y * width + z * width * height] = *(s_data_src + x);
            x += blockDim.x;
        }
        x = threadIdx.x;
    }
}

__global__ void g_filter_gauss_Y_flt(float *data, const size_t width, const size_t height, const size_t depth, const size_t radius, const size_t n_iter)
{
    size_t x = blockIdx.x;
    size_t y = threadIdx.y;

    extern __shared__ float s_data_GY_flt[];
    float *s_data_src = s_data_GY_flt + radius;
    float *s_data_dst = s_data_GY_flt + 2 * radius + height;

    if (y == 0) 
    {
        for (int i = radius; i--;)
        {
            *(s_data_GY_flt + i) = *(s_data_GY_flt + radius + height + i) = *(s_data_GY_flt + 2 * radius + 2 * height + i) = 0.0f;
        }
    }

    for (int z = 0; z < depth; ++z)
    {
        //inline size_t index = x + y * width + z * width * height;
        while(x < width && y < height)
        {
            *(s_data_src + y) = data[x + y * width + z * width * height];
            y += blockDim.y;
        }

        y = threadIdx.y;
        __syncthreads();

        for (int n = n_iter; n--;)
        {
            while(x < width && y < height)
            {
                *(s_data_dst + y) = *(s_data_src + y);
                for (int i = radius; i--;)
                {
                    *(s_data_dst + y) += *(s_data_src + y + (i + 1)) + *(s_data_src + y - (i + 1));
                }
                *(s_data_dst + y) /= 2 * radius + 1;
                y += blockDim.y;
            }

            y = threadIdx.y;
            float *tmp = s_data_src;
            s_data_src = s_data_dst;
            s_data_dst = tmp;
            __syncthreads();
        }

        while(x < width && y < height)
        {
            data[x + y * width + z * width * height] = *(s_data_src + y);
            y += blockDim.y;
        }
        y = threadIdx.y;
    }
}

__global__ void g_filter_boxcar_Z_flt(float *data, const size_t width, const size_t height, const size_t depth, const size_t radius)
{
    size_t x = blockIdx.x;
    size_t z = threadIdx.x;

    extern __shared__ float s_data_BZ_flt[];
    float *s_data_src = s_data_BZ_flt + radius;
    float *s_data_dst = s_data_BZ_flt + 2 * radius + depth;

    if (z == 0) 
    {
        for (int i = radius; i--;)
        {
            *(s_data_BZ_flt + i) = *(s_data_BZ_flt + radius + depth + i) = *(s_data_BZ_flt + 2 * radius + 2 * depth + i) = 0.0f;
        }
    }

    for (int y = 0; y < height; ++y)
    {
        //inline size_t index = x + y * width + z * width * height;
        while(x < width && z < depth)
        {
            *(s_data_src + z) = data[x + y * width + z * width * height];
            z += blockDim.x;
        }

        z = threadIdx.x;
        __syncthreads();

        while(x < width && z < depth)
        {
            *(s_data_dst + z) = *(s_data_src + z);
            for (int i = radius; i--;)
            {
                *(s_data_dst + z) += *(s_data_src + z + (i + 1)) + *(s_data_src + z - (i + 1));
            }
            *(s_data_dst + z) /= 2 * radius + 1;
            z += blockDim.x;
        }

        z = threadIdx.x;
        __syncthreads();

        while(x < width && z < depth)
        {
            data[x + y * width + z * width * height] = *(s_data_dst + z);
            z += blockDim.x;
        }
        z = threadIdx.x;
    }
}

__global__ void g_Mask8(float *data_box, char *maskData8, const size_t width, const size_t height, const size_t depth, const double threshold, float *rms_smooth, const int8_t value)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {return;}

    size_t index = x + y * width;

    while (index < width * height * depth)
    {
        if (fabs(data_box[index]) > threshold * (*rms_smooth)) {maskData8[index] = (char)1;}
        index += width * height;
    }
}

void GPU_DataCube_filter(char *data, char *originalData, int word_size, size_t data_size, size_t *axis_size, size_t radiusGauss, size_t n_iter, size_t radiusBoxcar)
{


    GPU_DataCube_filter_Chunked(data, originalData, word_size, data_size, axis_size, radiusGauss, n_iter, radiusBoxcar, 1);
}

void GPU_DataCube_filter_Chunked(char *data, char *originalData, int word_size, size_t data_size, size_t *axis_size, size_t radiusGauss, size_t n_iter, size_t radiusBoxcar, size_t number_of_chunks)
{
    if (!radiusGauss && ! radiusBoxcar) {return;}

    // Error at start?
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error at start: %s\n", hipGetErrorString(err));    
    }

    // check for CUDA capable device
    hipFree(0);
    int deviceCount;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        exit(0);
    }

    hipGetDeviceProperties(&prop, 0);

    // Allocate and copy Datacube data onto GPU
    float *d_data;
    float *d_data_box;

    size_t x_overlap = axis_size[1] * axis_size[2] * 2 * radiusGauss;
    size_t y_overlap = axis_size[0] * axis_size[2] * 2 * radiusGauss;
    size_t z_overlap = axis_size[0] * axis_size[1] * 2 * radiusBoxcar;

    size_t box_size =   (data_size 
                            + (x_overlap > y_overlap ? (x_overlap > z_overlap ? x_overlap : z_overlap) 
                            : (y_overlap > z_overlap ? y_overlap : z_overlap))
                        ) 
                        * sizeof(float);
    size_t slices_per_chunk = axis_size[2] / number_of_chunks;

    x_overlap = slices_per_chunk * axis_size[1] * 2 * radiusGauss;
    y_overlap = slices_per_chunk * axis_size[0] * 2 * radiusGauss;

    // if (prop.totalGlobalMem < 2 * box_size)
    // {
    //     number_of_chunks = ((2 * box_size) / prop.totalGlobalMem) + 1;
    //     slices_per_chunk /= number_of_chunks;
    //     slices_per_chunk++;
    // }

    // if (slices_per_chunk < 2 * radiusBoxcar + 1)
    // {
    //     printf("Insufficient memory on GPU to load enought slices of the cube to perform the boxcar filter.\n");
    //     exit(1);
    // }

    size_t chunk_overlap = x_overlap > y_overlap ? 
                        (x_overlap > z_overlap ? x_overlap : z_overlap) : 
                        (y_overlap > z_overlap ? y_overlap : z_overlap);

    err = hipMalloc((void**)&d_data, (slices_per_chunk + 2 * radiusBoxcar) * axis_size[0] * axis_size[1] * word_size * sizeof(char));
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
        exit(0);
    }

    err = hipMemset(d_data, 0, (slices_per_chunk + 2 * radiusBoxcar) * axis_size[0] * axis_size[1] * word_size * sizeof(char));
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
        exit(0);
    }

    err = hipMalloc((void**)&d_data_box, (slices_per_chunk * axis_size[0] * axis_size[1] + chunk_overlap) * sizeof(float));
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
        exit(0);
    }

    size_t remaining_slices = axis_size[2];
    int processed_chunks = 0;

    if (number_of_chunks > 1)
    {
        // TODO protect against thin cubes, where the first copy with a large boxcar filter may not succeed
        hipMemcpy(d_data + radiusBoxcar * axis_size[0] * axis_size[1], originalData, (slices_per_chunk + radiusBoxcar) * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToDevice);

        // Error after mem copy?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error at mem Copy to device: %s\n", hipGetErrorString(err));    
        }

        // Error before Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error before Xkernel launch: %s\n", hipGetErrorString(err));    
        }

        // Gauss Filter size in X Direction
        dim3 blockSizeX(ceil((float)32 / number_of_chunks),32);
        dim3 gridSizeX((slices_per_chunk + blockSizeX.x - 1) / blockSizeX.x ,
                    (axis_size[1] + blockSizeX.y - 1) / blockSizeX.y);

        if (radiusGauss && !radiusBoxcar) g_DataCube_gauss_filter_XDir<<<gridSizeX, blockSizeX, axis_size[0] * sizeof(float) + (axis_size[0] + 2 * radiusGauss) * sizeof(float)>>>(d_data + radiusBoxcar * axis_size[0] * axis_size[1], d_data_box, word_size, axis_size[0], axis_size[1], slices_per_chunk, radiusGauss, n_iter);

        hipDeviceSynchronize();

        // Error after Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after Xkernel launch: %s\n", hipGetErrorString(err));    
        }

        // Gauss Filter in Y Direction
        dim3 blockSizeY(ceil((float)16 / number_of_chunks),16);
        dim3 gridSizeY((slices_per_chunk + blockSizeY.x - 1) / blockSizeY.x,
                    (axis_size[0] + blockSizeY.y - 1) / blockSizeY.y);

        // Error before Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error before Ykernel launch: %s\n", hipGetErrorString(err));    
        }

        if (radiusGauss && !radiusBoxcar) g_DataCube_gauss_filter_YDir<<<gridSizeY, blockSizeY>>>(d_data + radiusBoxcar * axis_size[0] * axis_size[1], d_data_box, word_size, axis_size[0], axis_size[1], slices_per_chunk, radiusGauss, n_iter);

        hipDeviceSynchronize();

        // Error after Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after Ykernel launch: %s\n", hipGetErrorString(err));    
        }

        // Boxcar Filter in Z Direction
        dim3 blockSizeZ(32,32);
        dim3 gridSizeZ((axis_size[0] + blockSizeZ.x - 1) / blockSizeZ.x,
                    (axis_size[1] + blockSizeZ.y - 1) / blockSizeZ.y);

        if (radiusBoxcar) g_DataCube_boxcar_filter_flt<<<gridSizeZ, blockSizeZ>>>(d_data, originalData, d_data_box, word_size, processed_chunks * slices_per_chunk, axis_size[0], axis_size[1], slices_per_chunk, radiusBoxcar, 0);

        hipDeviceSynchronize();

        if (radiusGauss && !radiusBoxcar)
        {
            hipMemcpy(originalData, d_data + radiusBoxcar * axis_size[0] * axis_size[1], slices_per_chunk * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToDevice);
            // Error after backkcopy??
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("Cuda error after backcopy: %s\n", hipGetErrorString(err));
            }
        }

        hipMemcpy(data, d_data + radiusBoxcar * axis_size[0] * axis_size[1], slices_per_chunk * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToHost);
        // Error after backkcopy??
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after backcopy: %s\n", hipGetErrorString(err));
        }

        remaining_slices -= slices_per_chunk;
        processed_chunks++;
    }

    while(remaining_slices > slices_per_chunk)
    {
        //size_t remaining_slices = axis_size[2] - i * slices_per_chunk;
        size_t slices_to_copy = min(slices_per_chunk + radiusBoxcar, remaining_slices);

        hipMemcpy(d_data + radiusBoxcar * axis_size[0] * axis_size[1], originalData + processed_chunks * slices_per_chunk * axis_size[0] * axis_size[1] * word_size, slices_to_copy *  axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToDevice);

        // If there are not enought slices left at the end fill the overlap region with zeroes where neccessary
        if (slices_to_copy < (slices_per_chunk + radiusBoxcar))
        {
            float *zeroes = (float*)calloc((slices_per_chunk + radiusBoxcar - slices_to_copy) * axis_size[0] * axis_size[1], sizeof(float));
            //err = hipMemcpy(d_data + (radiusBoxcar + slices_to_copy) * axis_size[0] * axis_size[1], zeroes, (slices_per_chunk + radiusBoxcar - slices_to_copy) * axis_size[0] * axis_size[1] * sizeof(float), hipMemcpyHostToDevice);
            err = hipMemset(d_data + (radiusBoxcar + slices_to_copy) * axis_size[0] * axis_size[1], 0, (slices_per_chunk + radiusBoxcar - slices_to_copy) * axis_size[0] * axis_size[1] * word_size);
            if (err != hipSuccess)
            {
                printf("Cuda error at memSet on device: %s\n", hipGetErrorString(err));
            }
            hipDeviceSynchronize();
        }
        
        // Error after mem copy?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error at mem Copy to device: %s\n", hipGetErrorString(err));    
        }

        // Error before Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error before Xkernel launch: %s\n", hipGetErrorString(err));    
        }

        // Gauss Filter size in X Direction
        dim3 blockSizeX(ceil((float)32 / number_of_chunks),32);
        dim3 gridSizeX((slices_per_chunk + blockSizeX.x - 1) / blockSizeX.x ,
                    (axis_size[1] + blockSizeX.y - 1) / blockSizeX.y);

        if (radiusGauss && !radiusBoxcar) g_DataCube_gauss_filter_XDir<<<gridSizeX, blockSizeX, axis_size[0] * sizeof(float) + (axis_size[0] + 2 * radiusGauss) * sizeof(float)>>>(d_data + radiusBoxcar * axis_size[0] * axis_size[1], d_data_box, word_size, axis_size[0], axis_size[1], slices_per_chunk, radiusGauss, n_iter);

        hipDeviceSynchronize();

        // Error after Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after Xkernel launch: %s\n", hipGetErrorString(err));    
        }

        // Gauss Filter in Y Direction
        dim3 blockSizeY(ceil((float)16 / number_of_chunks),16);
        dim3 gridSizeY((slices_per_chunk + blockSizeY.x - 1) / blockSizeY.x,
                    (axis_size[0] + blockSizeY.y - 1) / blockSizeY.y);

        // Error before Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error before Ykernel launch: %s\n", hipGetErrorString(err));    
        }

        if (radiusGauss && !radiusBoxcar) g_DataCube_gauss_filter_YDir<<<gridSizeY, blockSizeY>>>(d_data + radiusBoxcar * axis_size[0] * axis_size[1], d_data_box, word_size, axis_size[0], axis_size[1], slices_per_chunk, radiusGauss, n_iter);

        hipDeviceSynchronize();

        // Error after Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after Ykernel launch: %s\n", hipGetErrorString(err));    
        }

        // Boxcar Filter in Z Direction
        dim3 blockSizeZ(32,32);
        dim3 gridSizeZ((axis_size[0] + blockSizeZ.x - 1) / blockSizeZ.x,
                    (axis_size[1] + blockSizeZ.y - 1) / blockSizeZ.y);

        if (radiusBoxcar) g_DataCube_boxcar_filter_flt<<<gridSizeZ, blockSizeZ>>>(d_data, originalData, d_data_box, word_size, processed_chunks * slices_per_chunk, axis_size[0], axis_size[1], slices_per_chunk, radiusBoxcar, 1);

        hipDeviceSynchronize();

        if (radiusGauss && !radiusBoxcar)
        {
            hipMemcpy(originalData, d_data + radiusBoxcar * axis_size[0] * axis_size[1], slices_per_chunk * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToDevice);
            // Error after backkcopy??
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("Cuda error after backcopy: %s\n", hipGetErrorString(err));
            }
        }

        hipMemcpy(data + processed_chunks * slices_per_chunk * axis_size[0] * axis_size[1] * word_size, d_data + radiusBoxcar * axis_size[0] * axis_size[1], slices_per_chunk * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToHost);
        // Error after backkcopy??
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after backcopy: %s\n", hipGetErrorString(err));
        }

        remaining_slices -= slices_per_chunk;

        processed_chunks++;
    }

    // err = hipMalloc((void**)&d_data, data_size * word_size * sizeof(char));
    // if (err != hipSuccess)
    // {
    //     printf("%s\n", hipGetErrorString(err));
    //     exit(0);
    // }


    // err = hipMalloc((void**)&d_data_box, (data_size + 
    //                                         (x_overlap > y_overlap ? (x_overlap > z_overlap ? x_overlap : z_overlap) 
    //                                         : (y_overlap > z_overlap ? y_overlap : z_overlap))) 
    //                                         * sizeof(float));
    // if (err != hipSuccess)
    // {
    //     printf("%s\n", hipGetErrorString(err));
    //     exit(0);
    // }

    // for (int i = 0; i < blockSizeX.x; i++)
    // {
    //     for (int j = 0; j < blockSizeX.y; j++)
    //     {
    //         hipMemcpy(d_data 
    //                         + min(i * axis_size[0] * axis_size[1] * gridSizeX.x + (gridSizeX.x - 1) * axis_size[0] * axis_size[1], (axis_size[2]- 1) * axis_size[0] * axis_size[1])
    //                         + min(j * axis_size[0] * gridSizeX.y + (gridSizeX.y - 1) * axis_size[0], (axis_size[1] - 1) * axis_size[0])
    //                         + axis_size[0] - 1, 
    //                         &flag, sizeof(char), hipMemcpyHostToDevice);
    //     }
    // }

    size_t last_chunk_size = remaining_slices;

    if (last_chunk_size > 0)
    {
        hipMemcpy(d_data + radiusBoxcar * axis_size[0] * axis_size[1], originalData + processed_chunks * slices_per_chunk * axis_size[0] * axis_size[1] * word_size, last_chunk_size * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToDevice);
        // Error after mem copy?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error at last mem Copy to device: %s\n", hipGetErrorString(err));    
        }

        // Error before Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error before Xkernel launch: %s\n", hipGetErrorString(err));    
        }

        // Gauss Filter size in X Direction
        dim3 blockSizeX(ceil((float)32 / number_of_chunks),32);
        dim3 gridSizeX((last_chunk_size + blockSizeX.x - 1) / blockSizeX.x ,
                    (axis_size[1] + blockSizeX.y - 1) / blockSizeX.y);

        if (radiusGauss && !radiusBoxcar) g_DataCube_gauss_filter_XDir<<<gridSizeX, blockSizeX, axis_size[0] * sizeof(float) + (axis_size[0] + 2 * radiusGauss) * sizeof(float)>>>(d_data + radiusBoxcar * axis_size[0] * axis_size[1], d_data_box, word_size, axis_size[0], axis_size[1], last_chunk_size, radiusGauss, n_iter);

        hipDeviceSynchronize();

        // Error after Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after Xkernel launch: %s\n", hipGetErrorString(err));    
        }

        // Gauss Filter in Y Direction
        dim3 blockSizeY(ceil((float)16 / number_of_chunks),16);
        dim3 gridSizeY((last_chunk_size + blockSizeY.x - 1) / blockSizeY.x,
                    (axis_size[0] + blockSizeY.y - 1) / blockSizeY.y);

        // Error before Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error before Ykernel launch: %s\n", hipGetErrorString(err));    
        }

        if (radiusGauss && !radiusBoxcar) g_DataCube_gauss_filter_YDir<<<gridSizeY, blockSizeY>>>(d_data + radiusBoxcar * axis_size[0] * axis_size[1], d_data_box, word_size, axis_size[0], axis_size[1], last_chunk_size, radiusGauss, n_iter);

        hipDeviceSynchronize();

        // Error after Kernel Launch?
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after Ykernel launch: %s\n", hipGetErrorString(err));    
        }

        // Boxcar Filter in Z Direction
        dim3 blockSizeZ(32,32);
        dim3 gridSizeZ((axis_size[0] + blockSizeZ.x - 1) / blockSizeZ.x,
                    (axis_size[1] + blockSizeZ.y - 1) / blockSizeZ.y);

        if (radiusBoxcar) g_DataCube_boxcar_filter_flt<<<gridSizeZ, blockSizeZ>>>(d_data, originalData, d_data_box, word_size, processed_chunks * slices_per_chunk, axis_size[0], axis_size[1], last_chunk_size, radiusBoxcar, 2);

        hipDeviceSynchronize();

        if (radiusGauss && !radiusBoxcar)
        {
            //g_DataCube_copy_back_smoothed_cube(originalData, d_data, word_size, axis_size[0], axis_size[1], last_chunk_size);
        }

        hipMemcpy(data + processed_chunks * slices_per_chunk * axis_size[0] * axis_size[1] * word_size, d_data + radiusBoxcar * axis_size[0] * axis_size[1], last_chunk_size * axis_size[0] * axis_size[1] * word_size * sizeof(char), hipMemcpyDeviceToHost);
        // Error after backkcopy??
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("Cuda error after backcopy: %s\n", hipGetErrorString(err));
        }
    }

    hipFree(d_data);
    hipFree(d_data_box);

    // Error after free mem??
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Cuda error after freeing memory: %s\n", hipGetErrorString(err));    
    }
}

void GPU_DataCube_copy_mask_8_to_1(char* maskData1, char* maskData8, const size_t *axis_size)
{
    dim3 blockSize(32,32);
    dim3 gridSize((ceil(axis_size[0] / 8.0f) + blockSize.x - 1) / blockSize.x,
                            (axis_size[1] + blockSize.y - 1) / blockSize.y);

    g_DataCube_copy_mask_8_to_1<<<gridSize, blockSize>>>(maskData1, maskData8, axis_size[0], axis_size[1], axis_size[2]);
    hipDeviceSynchronize();
}

__global__ void g_DataCube_boxcar_filter_flt(float *data, char *originalData, float *data_box, int word_size, const size_t startSlice, size_t width, size_t height, size_t depth, size_t radius, size_t chunck_type)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t jump = width * height;

    if (x < width && y < height)
    {
        data = data + (x + y * width);
        data_box = data_box + (x + y * width);

        d_filter_chunk_boxcar_1d_flt(data, originalData, data_box, startSlice, depth, radius, jump, chunck_type);
    }
}

__global__ void g_DataCube_gauss_filter_XDir(float *data, float *data_box, int word_size, size_t width, size_t height, size_t depth, size_t radius, size_t n_iter)
{
    size_t thread_count = blockDim.x * blockDim.y;
    size_t thread_index = threadIdx.x * blockDim.y + threadIdx.y;

    const size_t filter_size = 2 * radius + 1;
	const float inv_filter_size = 1.0 / filter_size;

    extern __shared__ float s_data[];
    float *s_data_box = &s_data[width];

    size_t start_index = blockIdx.x * blockDim.x * width * height + blockIdx.y * blockDim.y * width;    

    for (int iter = 0; iter < blockDim.x * blockDim.y; iter++)
    {
        if (blockIdx.y * blockDim.y + (iter % blockDim.y) >= height) continue;

        size_t data_index = start_index + (iter % blockDim.y) * width + (iter / blockDim.y) * width * height;

        if (data_index >= width * height * depth) continue;

        for (size_t i = 0; i < (float)width / thread_count; i++)
        {
            size_t j = thread_index + thread_count * i;
            if (j < width)
            {
                s_data[j] = s_data_box[radius + j] = data[data_index + j];
            }
        }

         for (int i = radius; i--;) s_data_box[i] = s_data_box[radius + width + i] = 0.0;

        __syncthreads();

        for (size_t k = n_iter; k--;)
        {
            for (int i = 0; i < (float)width / thread_count; i++)
            {
                int j = thread_index + thread_count * i;
                if (j < width)
                {
                    s_data[j] = 0.0;
                    for(int f = filter_size; f--;) s_data[j] += s_data_box[j + f];
                    s_data[j] *= inv_filter_size;
                }
            }

            __syncthreads();

            for (int i = 0; i < (float)width / thread_count; i++)
            {
                int j = thread_index + thread_count * i;
                if (j < width)
                {
                    s_data_box[radius + j] = s_data[j];
                }
            }

            __syncthreads();
        }

        for (int i = 0; i < width / (float)thread_count; i++)
        {
            int j = thread_index + thread_count * i;
            if (j < width)
            {
                data[data_index + j] = s_data[j];
            }
        }
    }
}

__global__ void g_DataCube_gauss_filter_YDir(float *data, float *data_box, int word_size, size_t width, size_t height, size_t depth, size_t radius, size_t n_iter)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < depth && y < width)
    {
        data = data + x * width * height + y;
        data_box = data_box + x * width * (height + 2 * radius) 
                            + y;

        for(size_t i = n_iter; i--;) d_filter_boxcar_1d_flt(data, data_box, height, radius, width);
    }
}

__global__ void g_DataCube_copy_mask_8_to_1(char* maskData1, char* maskData8, size_t width, size_t height, size_t depth)
{
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t z = 0;

    size_t jump = width * height;
    size_t jump1 = ((width + 7) / 8) * height;

    while (z < depth)
    {
        size_t x = (blockIdx.x * blockDim.x + threadIdx.x) * 8;

        if (x < width && y < height)
        {
            size_t indexSrc = width * y + x + z * jump;
            size_t indexDst = ((width + 7) / 8) * y + x / 8 + z * jump1;
            int8_t *srcPtr = (int8_t*)maskData8 + indexSrc;
            char *dstPtr = maskData1 + indexDst;

            u_int8_t result = 0;

            result |= (*srcPtr++ != 0) << 7;
            result |= (*srcPtr++ != 0) << 6;
            result |= (*srcPtr++ != 0) << 5;
            result |= (*srcPtr++ != 0) << 4;
            result |= (*srcPtr++ != 0) << 3;
            result |= (*srcPtr++ != 0) << 2;
            result |= (*srcPtr++ != 0) << 1;
            result |= (*srcPtr != 0) << 0;

            *dstPtr = (char)result;

            //x += blockDim.x * 8;
        }

        z++;
    }
}

__global__ void g_DataCube_copy_back_smoothed_cube(char *originalData, float *data, int word_size, size_t width, size_t height, size_t depth)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t jump = width * height;

    if (x < width && y < height)
    {
        size_t z = 0;
        size_t index = x + y * width;
        while (z < depth)
        {
            originalData[index + z * jump] = IS_NAN(originalData[index + z * jump]) ? originalData[index + z * jump] : data[index + z * jump];
            z++;
        }
    }
}

__global__ void g_DataCube_stat_mad_flt(float *data, float *data_box, size_t width, size_t height, size_t depth, const float value, const size_t cadence, const int range)
{
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t index = y * blockDim.x * gridDim.x + x;
    const size_t local_index = y * blockDim.x + x;

    const size_t thread_count = blockDim.x * blockDim.y;
    const size_t thread_count_total = gridDim.x * gridDim.y * thread_count;
    const size_t max_medians_per_thread = ((((width * height * depth + cadence - 1) / cadence + thread_count_total - 1) / thread_count_total) + 4) / 5;

    float *ptr = data + width * height * depth - 1 - index * cadence;
    float *ptr_box_start = data_box + index * (max_medians_per_thread + 1);
    float *ptr_box = ptr_box_start + 1;

    extern __shared__ float s_data_mad[];
    float *s_data_start = s_data_mad + local_index * 6;
    float *s_data_median_start = s_data_mad + thread_count * 6 + local_index;
    int counter = 0;

    if (range == 0)
    {
        while (ptr >= data)
        {
            while (ptr >= data && counter < 5)
            {
                if (IS_NOT_NAN(*ptr))
                {
                    *(s_data_start + counter) = fabs(*ptr - value);
                    counter++;
                }
                ptr -= thread_count_total * cadence;
            }

            *(s_data_start + 5) = 0;
            
            if (counter > 0)
            {
                d_sort_arr_flt(s_data_start, counter);
                *s_data_median_start = counter % 2 != 0 ? 
                                        *(s_data_start + counter / 2) : 
                                        (*(s_data_start + counter / 2) + *(s_data_start + (counter / 2 - 1))) / 2;
                *(s_data_start + 5) = 1;
            }
            __syncthreads();

            counter = 2;
            while (local_index % counter == 0 && counter <= thread_count)
            {
                *(s_data_start + 5) = *(s_data_start + 5) + *(s_data_start + 5 + 6 * counter / 2);
                counter *= 2;
            }

            counter = 0;

            __syncthreads();

            if (local_index == 0)
            {
                d_sort_arr_flt(s_data_median_start, (int)*(s_data_start + 5));
                // data_box[(int)atomicAdd(data_box + width * height * depth - 1, 1)] = *s_data_median_start;
                //thrust::sort(s_data_median_start, s_data_median_start + (int)*s_data_start);
                data_box[(int)atomicAdd(data_box + width * height * depth - 1, 1)] = (int)*(s_data_start + 5) % 2 != 0 ? 
                                                                                    *(s_data_median_start + (int)*(s_data_start + 5) / 2) :
                                                                                    (*(s_data_median_start + (int)*(s_data_start + 5) / 2) + *(s_data_median_start + (int)*(s_data_start + 5) / 2 - 1)) / 2;
            }
            __syncthreads();
        }
    }
}

__global__ void g_DataCube_stat_mad_flt_2(float *data, float *data_box, size_t size, const float value, const size_t cadence, const int range, const float pivot)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_count = blockDim.x * gridDim.x;

    extern __shared__ float s_data_stat_mad[];
}

__global__ void g_std_dev_val_flt(float *data, float *data_dst_duo, const size_t size, const float value, const size_t cadence, const int range)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_count = blockDim.x * gridDim.x;

    extern __shared__ float s_data_sdf[];
    float *s_data_sdf_start = s_data_sdf + threadIdx.x * 2;
    *s_data_sdf_start = 0.0f;
    *(s_data_sdf_start + 1) = 0.0f;

    const float *ptr = data + size + index * cadence;
    const float *ptr2 = data + cadence * thread_count - 1;

    while (ptr > ptr2)
    {
        ptr -= cadence * thread_count;

        if((range == 0 && IS_NOT_NAN(*ptr)) || (range < 0 && *ptr < 0.0) || (range > 0 && *ptr > 0.0))
		{
			*s_data_sdf_start += (*ptr - value) * (*ptr - value);
			++*(s_data_sdf_start + 1);
		}
    }

    __syncthreads();

    int counter = 2;
    while (counter / 2 < blockDim.x)
    {
        if (threadIdx.x % counter == 0)
        {
            if (*(s_data_sdf_start + counter) < 0.0f) {atomicAdd(data_dst_duo + 2, 1);}
            *s_data_sdf_start += *(s_data_sdf_start + counter);
            *(s_data_sdf_start + 1) += *(s_data_sdf_start + 1 + counter);
        }
        counter *= 2;
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        atomicAdd(data_dst_duo, *s_data_sdf_start);
        atomicAdd(data_dst_duo + 1, *(s_data_sdf_start + 1));
    }
}

__global__ void g_std_dev_val_flt_final_step(float *data_duo)
{
    *data_duo = sqrt(*data_duo / *(data_duo + 1));
}

__device__ void d_filter_boxcar_1d_flt(float *data, float *data_copy, const size_t size, const size_t filter_radius, const size_t jump)
{
    // Define filter size
	const size_t filter_size = 2 * filter_radius + 1;
	const float inv_filter_size = 1.0 / filter_size;
	size_t i;

	// Make copy of data, taking care of NaN
	for(i = size; i--;) data_copy[(filter_radius + i) * jump] = FILTER_NAN(data[i * jump]);
	
	// Fill overlap regions with 0
	for(i = filter_radius; i--;) data_copy[i * jump] = data_copy[(size + filter_radius + i) * jump] = 0.0;
	
	// Apply boxcar filter to last data point
	data[(size - 1) * jump] = 0.0;
	for(i = filter_size; i--;) data[(size - 1) * jump] += data_copy[(size + i - 1) * jump];
	data[(size - 1) * jump] *= inv_filter_size;
	
	// Recursively apply boxcar filter to all previous data points
	for(i = size - 1; i--;) data[i * jump] = data[(i + 1) * jump] + (data_copy[i * jump] - data_copy[(filter_size + i) * jump]) * inv_filter_size;
	
	return;
}

__device__ void d_filter_chunk_boxcar_1d_flt(float *data, char *originalData, float *data_copy, const size_t startSlice, const size_t size, const size_t filter_radius, const size_t jump, size_t chunk_type)
{
    // Define filter size
	const size_t filter_size = 2 * filter_radius + 1;
	const float inv_filter_size = 1.0 / filter_size;
	size_t i;

    if (chunk_type != 2)
    {
        for(i = filter_radius; i--;) data_copy[i * jump] = FILTER_NAN(data[i * jump]);
        for(i = filter_radius; i--;) data_copy[(size + filter_radius + i) * jump] = FILTER_NAN(data[(size + filter_radius + i) * jump]);
    }
    else
    {
        for(i = filter_radius; i--;) data_copy[i * jump] = FILTER_NAN(data[i * jump]);
        for(i = filter_radius; i--;) data_copy[(size + filter_radius + i) * jump] = 0.0;
    }

    // // Fill overlap regions
    // if (chunk_type == 0)
    // {
    //     for(i = filter_radius; i--;) data_copy[i * jump] = 0.0;
    //     for(i = filter_radius; i--;) data_copy[(size + filter_radius + i) * jump] = FILTER_NAN(data[(size + filter_radius + i) * jump]);
    // }
    // else if (chunk_type == 1)
    // {
    //     for(i = filter_radius; i--;) data_copy[i * jump] = FILTER_NAN(data[i * jump]);
    //     for(i = filter_radius; i--;) data_copy[(size + filter_radius + i) * jump] = FILTER_NAN(data[(size + filter_radius + i) * jump]);
    // }
    // else if (chunk_type == 2)
    // {
    //     for(i = filter_radius; i--;) data_copy[i * jump] = FILTER_NAN(data[i * jump]);
    //     for(i = filter_radius; i--;) data_copy[(size + filter_radius + i) * jump] = 0.0;
    // }

    // Write elements at the end of the data chunk back to the front end overlap for next chunk
    if (chunk_type != 2)
    {
        for(i = 0; i < filter_radius; i++) data[i * jump] = data[(size + i) * jump];
    }

	// Make copy of data, taking care of NaN
	for(i = size; i--;) data_copy[(filter_radius + i) * jump] = FILTER_NAN(data[(filter_radius + i) * jump]);
	
	// Apply boxcar filter to last data point
	data[(size + filter_radius - 1) * jump] = 0.0;
	for(i = filter_size; i--;) data[(size + filter_radius - 1) * jump] += data_copy[(size + i - 1) * jump];
	data[(size + filter_radius - 1) * jump] *= inv_filter_size;
	
	// Recursively apply boxcar filter to all previous data points
	for(i = size - 1; i--;) data[(filter_radius + i) * jump] = data[(filter_radius + i + 1) * jump] + (data_copy[i * jump] - data_copy[(filter_size + i) * jump]) * inv_filter_size;
	
	return;
}

__device__ void d_sort_arr_flt(float *arr, size_t size)
{
    float tmp;
    for (int i = size + 1; --i;)
    {
        int j = size - 1;
        while(j > size - i)
        {
            tmp = arr[j - 1];
            if (arr[j - 1] > arr[j])
            {
                arr[j - 1] = arr[j];
                arr[j] = tmp;
            }
            --j;
        }
    }
}

void sort_arr_flt(float *arr, size_t size)
{
    float tmp;
    for (int i = size + 1; --i;)
    {
        int j = size - 1;
        while(j > size - i)
        {
            tmp = arr[j - 1];
            if (arr[j - 1] > arr[j])
            {
                arr[j - 1] = arr[j];
                arr[j] = tmp;
            }
            --j;
        }
    }
}